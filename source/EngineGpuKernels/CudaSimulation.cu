#include "hip/hip_runtime.h"
#include "CudaSimulation.cuh"

#include <functional>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <>
#include <cuda/hip/hip_runtime_api.h>

#include "Base/Exceptions.h"
#include "EngineInterface/InspectedEntityIds.h"
#include "EngineInterface/SimulationParameters.h"
#include "EngineInterface/GpuSettings.h"

#include "Base/LoggingService.h"
#include "Base/ServiceLocator.h"
#include "AccessKernels.cuh"
#include "AccessTOs.cuh"
#include "Base.cuh"
#include "GarbageCollectorKernels.cuh"
#include "ConstantMemory.cuh"
#include "CudaMemoryManager.cuh"
#include "CudaMonitorData.cuh"
#include "Entities.cuh"
#include "Map.cuh"
#include "MonitorKernels.cuh"
#include "EditKernels.cuh"
#include "RenderingKernels.cuh"
#include "SimulationData.cuh"
#include "SimulationKernelLauncher.cuh"
#include "SimulationResult.cuh"
#include "SelectionResult.cuh"
#include "RenderingData.cuh"

namespace
{
    class CudaInitializer
    {
    public:
        static void init() { [[maybe_unused]] static CudaInitializer instance; }

        CudaInitializer()
        {
            int deviceNumber = getDeviceNumberOfHighestComputeCapability();

            auto loggingService = ServiceLocator::getInstance().getService<LoggingService>();
            auto result = hipSetDevice(deviceNumber);
            if (result != hipSuccess) {
                throw SystemRequirementNotMetException("CUDA device could not be initialized.");
            }

            std::stringstream stream;
            stream << "device " << deviceNumber << " is set";
            loggingService->logMessage(Priority::Important, stream.str());
        }

        ~CudaInitializer() { hipDeviceReset(); }

    private:
        int getDeviceNumberOfHighestComputeCapability()
        {
            auto loggingService = ServiceLocator::getInstance().getService<LoggingService>();
            int result = 0;
            int numberOfDevices;
            CHECK_FOR_CUDA_ERROR(hipGetDeviceCount(&numberOfDevices));
            if (numberOfDevices < 1) {
                throw SystemRequirementNotMetException("No CUDA device found.");
            }
            {
                std::stringstream stream;
                if (1 == numberOfDevices) {
                    stream << "1 CUDA device found";
                } else {
                    stream << numberOfDevices << " CUDA devices found";
                }
                loggingService->logMessage(Priority::Important, stream.str());
            }

            int highestComputeCapability = 0;
            for (int deviceNumber = 0; deviceNumber < numberOfDevices; ++deviceNumber) {
                hipDeviceProp_t prop;
                CHECK_FOR_CUDA_ERROR(hipGetDeviceProperties(&prop, deviceNumber));

                std::stringstream stream;
                stream << "device " << deviceNumber << ": " << prop.name << " with compute capability " << prop.major
                       << "." << prop.minor;
                loggingService->logMessage(Priority::Important, stream.str());

                int computeCapability = prop.major * 100 + prop.minor;
                if (computeCapability > highestComputeCapability) {
                    result = deviceNumber;
                    highestComputeCapability = computeCapability;
                }
            }
            if (highestComputeCapability < 600) {
                throw SystemRequirementNotMetException(
                    "No CUDA device with compute capability of 6.0 or higher found.");
            }

            return result;
        }
    };
}

void _CudaSimulation::initCuda()
{
    CudaInitializer::init();
}

_CudaSimulation::_CudaSimulation(uint64_t timestep, Settings const& settings, GpuSettings const& gpuSettings)
{
    CHECK_FOR_CUDA_ERROR(hipGetLastError());

    setSimulationParameters(settings.simulationParameters);
    setSimulationParametersSpots(settings.simulationParametersSpots);
    setGpuConstants(gpuSettings);
    setFlowFieldSettings(settings.flowFieldSettings);

    auto loggingService = ServiceLocator::getInstance().getService<LoggingService>();
    loggingService->logMessage(Priority::Important, "initialize simulation");

    _currentTimestep.store(timestep);
    _cudaSimulationData = new SimulationData();
    _cudaRenderingData = new RenderingData();
    _cudaSimulationResult = new SimulationResult();
    _cudaSelectionResult = new SelectionResult();
    _cudaAccessTO = new DataAccessTO();
    _cudaMonitorData = new CudaMonitorData();

    _simulationKernels = new SimulationKernelLauncher();

    int2 worldSize{settings.generalSettings.worldSizeX, settings.generalSettings.worldSizeY};
    _cudaSimulationData->init(worldSize);
    _cudaRenderingData->init();
    _cudaMonitorData->init();
    _cudaSimulationResult->init();
    _cudaSelectionResult->init();

    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numCells);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numParticles);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numTokens);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numStringBytes);
    CudaMemoryManager::getInstance().acquireMemory<char>(Const::MetadataMemorySize, _cudaAccessTO->stringBytes);

    //default array sizes for empty simulation (will be resized later if not sufficient)
    resizeArrays({100000, 100000, 10000});
}

_CudaSimulation::~_CudaSimulation()
{
    _cudaSimulationData->free();
    _cudaRenderingData->free();
    _cudaMonitorData->free();
    _cudaSimulationResult->free();
    _cudaSelectionResult->free();

    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->cells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->particles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->tokens);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->stringBytes);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numCells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numParticles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numTokens);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numStringBytes);

    auto loggingService = ServiceLocator::getInstance().getService<LoggingService>();
    loggingService->logMessage(Priority::Important, "close simulation");

    delete _cudaAccessTO;
    delete _cudaSimulationData;
    delete _cudaRenderingData;
    delete _cudaMonitorData;
    delete _simulationKernels;
}

void* _CudaSimulation::registerImageResource(GLuint image)
{
    hipGraphicsResource* cudaResource;

    CHECK_FOR_CUDA_ERROR(
        hipGraphicsGLRegisterImage(&cudaResource, image, GL_TEXTURE_2D, cudaGraphicsMapFlagsReadOnly));

    return reinterpret_cast<void*>(cudaResource);
}

void _CudaSimulation::calcTimestep()
{
    _simulationKernels->calcTimestep(_gpuSettings, *_cudaSimulationData, *_cudaSimulationResult);
    automaticResizeArrays();
    ++_currentTimestep;
}

void _CudaSimulation::drawVectorGraphics(
    float2 const& rectUpperLeft,
    float2 const& rectLowerRight,
    void* cudaResource,
    int2 const& imageSize,
    double zoom)
{
    auto cudaResourceImpl = reinterpret_cast<hipGraphicsResource*>(cudaResource);
    CHECK_FOR_CUDA_ERROR(hipGraphicsMapResources(1, &cudaResourceImpl));

    hipArray* mappedArray;
    CHECK_FOR_CUDA_ERROR(hipGraphicsSubResourceGetMappedArray(&mappedArray, cudaResourceImpl, 0, 0));

    _cudaRenderingData->resizeImageIfNecessary(imageSize);

    DEPRECATED_KERNEL_CALL_HOST_SYNC(
        drawImageKernel,
        rectUpperLeft,
        rectLowerRight,
        imageSize,
        static_cast<float>(zoom),
        *_cudaSimulationData,
        *_cudaRenderingData);

    const size_t widthBytes = sizeof(uint64_t) * imageSize.x;
    CHECK_FOR_CUDA_ERROR(hipMemcpy2DToArray(
        mappedArray,
        0,
        0,
        _cudaRenderingData->imageData,
        widthBytes,
        widthBytes,
        imageSize.y,
        hipMemcpyDeviceToDevice));

    CHECK_FOR_CUDA_ERROR(hipGraphicsUnmapResources(1, &cudaResourceImpl));
}

void _CudaSimulation::getSimulationData(
    int2 const& rectUpperLeft,
    int2 const& rectLowerRight,
    DataAccessTO const& dataTO)
{
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaGetSimulationData, rectUpperLeft, rectLowerRight, *_cudaSimulationData, *_cudaAccessTO);
    copyFromGpu(dataTO);
}

void _CudaSimulation::getSelectedSimulationData(bool includeClusters, DataAccessTO const& dataTO)
{
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaGetSelectedSimulationData, *_cudaSimulationData, includeClusters, * _cudaAccessTO);
    copyFromGpu(dataTO);
}

void _CudaSimulation::getInspectedSimulationData(std::vector<uint64_t> entityIds, DataAccessTO const& dataTO)
{
    InspectedEntityIds ids;
    if (entityIds.size() > Const::MaxInspectedEntities) {
        return;
    }
    for (int i = 0; i < entityIds.size(); ++i) {
        ids.values[i] = entityIds.at(i);
    }
    if (entityIds.size() < Const::MaxInspectedEntities) {
        ids.values[entityIds.size()] = 0;
    }
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaGetInspectedSimulationData, *_cudaSimulationData, ids, *_cudaAccessTO);
    copyFromGpu(dataTO);
}

void _CudaSimulation::getOverlayData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataAccessTO const& dataTO)
{
    DEPRECATED_KERNEL_CALL_HOST_SYNC(
        cudaGetSimulationOverlayData, rectUpperLeft, rectLowerRight, *_cudaSimulationData, *_cudaAccessTO);
    CHECK_FOR_CUDA_ERROR(hipMemcpy(dataTO.numCells, _cudaAccessTO->numCells, sizeof(int), hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        dataTO.cells, _cudaAccessTO->cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(
        hipMemcpy(dataTO.numParticles, _cudaAccessTO->numParticles, sizeof(int), hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        dataTO.particles,
        _cudaAccessTO->particles,
        sizeof(CellAccessTO) * (*dataTO.numParticles),
        hipMemcpyDeviceToHost));
}

void _CudaSimulation::addAndSelectSimulationData(DataAccessTO const& dataTO)
{
    copyToGpu(dataTO);
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaRemoveSelection, *_cudaSimulationData);
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaSetSimulationAccessData, *_cudaSimulationData, *_cudaAccessTO, true);
}

void _CudaSimulation::setSimulationData(DataAccessTO const& dataTO)
{
    copyToGpu(dataTO);
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaClearData, *_cudaSimulationData);
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaSetSimulationAccessData, *_cudaSimulationData, *_cudaAccessTO, false);
}

void _CudaSimulation::removeSelectedEntities(bool includeClusters)
{
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaRemoveSelectedEntities, *_cudaSimulationData, includeClusters);
}

void _CudaSimulation::changeInspectedSimulationData(DataAccessTO const& changeDataTO)
{
    copyToGpu(changeDataTO);
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaChangeSimulationData, *_cudaSimulationData, *_cudaAccessTO);
}

void _CudaSimulation::applyForce(ApplyForceData const& applyData)
{
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaApplyForce, applyData, *_cudaSimulationData);
}

void _CudaSimulation::switchSelection(PointSelectionData const& pointData)
{
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaSwitchSelection, pointData, *_cudaSimulationData);
}

void _CudaSimulation::swapSelection(PointSelectionData const& pointData)
{
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaSwapSelection, pointData, *_cudaSimulationData);
}

void _CudaSimulation::setSelection(AreaSelectionData const& selectionData)
{
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaSetSelection, selectionData, *_cudaSimulationData);
}

 SelectionShallowData _CudaSimulation::getSelectionShallowData()
{
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaGetSelectionShallowData, *_cudaSimulationData, *_cudaSelectionResult);
    return _cudaSelectionResult->getSelectionShallowData();
}

void _CudaSimulation::shallowUpdateSelectedEntities(ShallowUpdateSelectionData const& shallowUpdateData)
{
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaShallowUpdateSelectedEntities, shallowUpdateData, *_cudaSimulationData);
}

void _CudaSimulation::removeSelection()
{
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaRemoveSelection, *_cudaSimulationData);
}

void _CudaSimulation::updateSelection()
{
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaUpdateSelection, *_cudaSimulationData);
}

void _CudaSimulation::colorSelectedEntities(unsigned char color, bool includeClusters)
{
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaColorSelectedEntities, *_cudaSimulationData, color, includeClusters);
}

void _CudaSimulation::setGpuConstants(GpuSettings const& gpuConstants)
{
    _gpuSettings = gpuConstants;

    CHECK_FOR_CUDA_ERROR(
        hipMemcpyToSymbol(HIP_SYMBOL(cudaThreadSettings), &gpuConstants, sizeof(GpuSettings), 0, hipMemcpyHostToDevice));
}

auto _CudaSimulation::getArraySizes() const -> ArraySizes
{
    return {
        _cudaSimulationData->entities.cells.getSize_host(),
        _cudaSimulationData->entities.particles.getSize_host(),
        _cudaSimulationData->entities.tokens.getSize_host()};
}

OverallStatistics _CudaSimulation::getMonitorData()
{
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaGetCudaMonitorData, *_cudaSimulationData, *_cudaMonitorData);
    
    OverallStatistics result;

    auto monitorData = _cudaMonitorData->getMonitorData(getCurrentTimestep());
    result.timeStep = monitorData.timeStep;
    result.numCells = monitorData.numCells;
    result.numParticles = monitorData.numParticles;
    result.numTokens = monitorData.numTokens;
    result.totalInternalEnergy = monitorData.totalInternalEnergy;

    auto processStatistics = _cudaSimulationResult->getStatistics();
    result.numCreatedCells = processStatistics.createdCells;
    result.numSuccessfulAttacks = processStatistics.sucessfulAttacks;
    result.numFailedAttacks = processStatistics.failedAttacks;
    result.numMuscleActivities = processStatistics.muscleActivities;
    return result;
}

uint64_t _CudaSimulation::getCurrentTimestep() const
{
    return _currentTimestep.load();
}

void _CudaSimulation::setCurrentTimestep(uint64_t timestep)
{
    _currentTimestep.store(timestep);
}

void _CudaSimulation::setSimulationParameters(SimulationParameters const& parameters)
{
    CHECK_FOR_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(
        cudaSimulationParameters), &parameters, sizeof(SimulationParameters), 0, hipMemcpyHostToDevice));
}

void _CudaSimulation::setSimulationParametersSpots(SimulationParametersSpots const& spots)
{
    CHECK_FOR_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(
        cudaSimulationParametersSpots), &spots, sizeof(SimulationParametersSpots), 0, hipMemcpyHostToDevice));
}

void _CudaSimulation::setFlowFieldSettings(FlowFieldSettings const& settings)
{
    CHECK_FOR_CUDA_ERROR(
        hipMemcpyToSymbol(HIP_SYMBOL(cudaFlowFieldSettings), &settings, sizeof(FlowFieldSettings), 0, hipMemcpyHostToDevice));
}


void _CudaSimulation::clear()
{
    DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaClearData, *_cudaSimulationData);
}

void _CudaSimulation::resizeArraysIfNecessary(ArraySizes const& additionals)
{
    if (_cudaSimulationData->shouldResize(
            additionals.cellArraySize, additionals.particleArraySize, additionals.tokenArraySize)) {
        resizeArrays(additionals);
    }
}

void _CudaSimulation::copyToGpu(DataAccessTO const& dataTO)
{
    CHECK_FOR_CUDA_ERROR(hipMemcpy(_cudaAccessTO->numCells, dataTO.numCells, sizeof(int), hipMemcpyHostToDevice));
    CHECK_FOR_CUDA_ERROR(
        hipMemcpy(_cudaAccessTO->numParticles, dataTO.numParticles, sizeof(int), hipMemcpyHostToDevice));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(_cudaAccessTO->numTokens, dataTO.numTokens, sizeof(int), hipMemcpyHostToDevice));
    CHECK_FOR_CUDA_ERROR(
        hipMemcpy(_cudaAccessTO->numStringBytes, dataTO.numStringBytes, sizeof(int), hipMemcpyHostToDevice));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        _cudaAccessTO->cells, dataTO.cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyHostToDevice));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        _cudaAccessTO->particles,
        dataTO.particles,
        sizeof(ParticleAccessTO) * (*dataTO.numParticles),
        hipMemcpyHostToDevice));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        _cudaAccessTO->tokens, dataTO.tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyHostToDevice));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        _cudaAccessTO->stringBytes,
        dataTO.stringBytes,
        sizeof(char) * (*dataTO.numStringBytes),
        hipMemcpyHostToDevice));
}

void _CudaSimulation::copyFromGpu(DataAccessTO const& dataTO)
{
    CHECK_FOR_CUDA_ERROR(hipMemcpy(dataTO.numCells, _cudaAccessTO->numCells, sizeof(int), hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(
        hipMemcpy(dataTO.numParticles, _cudaAccessTO->numParticles, sizeof(int), hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(dataTO.numTokens, _cudaAccessTO->numTokens, sizeof(int), hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(
        hipMemcpy(dataTO.numStringBytes, _cudaAccessTO->numStringBytes, sizeof(int), hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        dataTO.cells, _cudaAccessTO->cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        dataTO.particles,
        _cudaAccessTO->particles,
        sizeof(ParticleAccessTO) * (*dataTO.numParticles),
        hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        dataTO.tokens, _cudaAccessTO->tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        dataTO.stringBytes,
        _cudaAccessTO->stringBytes,
        sizeof(char) * (*dataTO.numStringBytes),
        hipMemcpyDeviceToHost));
}

void _CudaSimulation::automaticResizeArrays()
{
    //make check after every 10th time step
    if (_currentTimestep.load() % 10 == 0) {
        if (_cudaSimulationResult->isArrayResizeNeeded()) {
            resizeArrays({0, 0, 0});
        }
    }
}

void _CudaSimulation::resizeArrays(ArraySizes const& additionals)
{
    auto loggingService = ServiceLocator::getInstance().getService<LoggingService>();
    loggingService->logMessage(Priority::Important, "resize arrays");

    _cudaSimulationData->resizeEntitiesForCleanup(
        additionals.cellArraySize, additionals.particleArraySize, additionals.tokenArraySize);
    if (!_cudaSimulationData->isEmpty()) {
        DEPRECATED_KERNEL_CALL_HOST_SYNC(cudaCopyEntities, *_cudaSimulationData);
        _cudaSimulationData->resizeRemainings();
        _cudaSimulationData->swap();
    } else {
        _cudaSimulationData->resizeRemainings();
    }

    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->cells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->particles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->tokens);

    auto cellArraySize = _cudaSimulationData->entities.cells.getSize_host();
    auto tokenArraySize = _cudaSimulationData->entities.tokens.getSize_host();
    CudaMemoryManager::getInstance().acquireMemory<CellAccessTO>(cellArraySize, _cudaAccessTO->cells);
    CudaMemoryManager::getInstance().acquireMemory<ParticleAccessTO>(cellArraySize, _cudaAccessTO->particles);
    CudaMemoryManager::getInstance().acquireMemory<TokenAccessTO>(tokenArraySize, _cudaAccessTO->tokens);

    CHECK_FOR_CUDA_ERROR(hipGetLastError());

    loggingService->logMessage(Priority::Unimportant, "cell array size: " + std::to_string(cellArraySize));
    loggingService->logMessage(Priority::Unimportant, "particle array size: " + std::to_string(cellArraySize));
    loggingService->logMessage(Priority::Unimportant, "token array size: " + std::to_string(tokenArraySize));

        auto const memorySizeAfter = CudaMemoryManager::getInstance().getSizeOfAcquiredMemory();
    loggingService->logMessage(Priority::Important, std::to_string(memorySizeAfter / (1024 * 1024)) + " MB GPU memory acquired");
}
