#include "hip/hip_runtime.h"
﻿#include "DataAccessKernelLauncher.cuh"

/************************************************************************/
/* Implementation                                                       */
/************************************************************************/
void DataAccessKernelLauncher::getData(
    GpuSettings const& gpuSettings,
    SimulationData const& simulationData,
    int2 const& rectUpperLeft,
    int2 const& rectLowerRight,
    DataAccessTO const& dataTO)
{
    KERNEL_CALL_1_1(clearDataTO, dataTO);
    KERNEL_CALL(getCellDataWithoutConnections, rectUpperLeft, rectLowerRight, simulationData, dataTO);
    KERNEL_CALL(resolveConnections, simulationData, dataTO);
    KERNEL_CALL(getTokenData, simulationData, dataTO);
    KERNEL_CALL(getParticleData, rectUpperLeft, rectLowerRight, simulationData, dataTO);

    hipDeviceSynchronize();
    CHECK_FOR_CUDA_ERROR(hipGetLastError());
}

void DataAccessKernelLauncher::addData(GpuSettings const& gpuSettings, SimulationData data, DataAccessTO dataTO, bool selectData)
{
    KERNEL_CALL_1_1(prepareSetData, data);
    KERNEL_CALL(adaptNumberGenerator, data.numberGen, dataTO);
    KERNEL_CALL(createDataFromTO, data, dataTO, selectData);
    _garbageCollector.cleanupAfterDataManipulation(gpuSettings, data);
    if (selectData) {
        KERNEL_CALL_1_1(rolloutSelection, data);
    }

    hipDeviceSynchronize();
    CHECK_FOR_CUDA_ERROR(hipGetLastError());
}

void DataAccessKernelLauncher::clearData(GpuSettings const& gpuSettings, SimulationData data) {}
