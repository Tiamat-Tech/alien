#include "hip/hip_runtime.h"
﻿#include "DataAccessKernelsLauncher.cuh"

/************************************************************************/
/* Implementation                                                       */
/************************************************************************/
void DataAccessKernelsLauncher::getData(
    GpuSettings const& gpuSettings,
    SimulationData const& simulationData,
    int2 const& rectUpperLeft,
    int2 const& rectLowerRight,
    DataAccessTO const& dataTO)
{
    KERNEL_CALL_1_1(clearDataTO, dataTO);
    KERNEL_CALL(getCellDataWithoutConnections, rectUpperLeft, rectLowerRight, simulationData, dataTO);
    KERNEL_CALL(resolveConnections, simulationData, dataTO);
    KERNEL_CALL(getTokenData, simulationData, dataTO);
    KERNEL_CALL(getParticleData, rectUpperLeft, rectLowerRight, simulationData, dataTO);

    hipDeviceSynchronize();
    CHECK_FOR_CUDA_ERROR(hipGetLastError());
}

void DataAccessKernelsLauncher::addData(GpuSettings const& gpuSettings, SimulationData data, DataAccessTO dataTO, bool selectData)
{
    KERNEL_CALL_1_1(prepareSetData, data);
    KERNEL_CALL(adaptNumberGenerator, data.numberGen, dataTO);
    KERNEL_CALL(createDataFromTO, data, dataTO, selectData);
    _garbageCollector.cleanupAfterDataManipulation(gpuSettings, data);
    if (selectData) {
        KERNEL_CALL_1_1(cudaRolloutSelection, data);
    }

    hipDeviceSynchronize();
    CHECK_FOR_CUDA_ERROR(hipGetLastError());
}

void DataAccessKernelsLauncher::clearData(GpuSettings const& gpuSettings, SimulationData data) {}
