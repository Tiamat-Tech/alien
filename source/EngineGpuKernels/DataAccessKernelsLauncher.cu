#include "hip/hip_runtime.h"
﻿#include "DataAccessKernelsLauncher.cuh"

/************************************************************************/
/* Implementation                                                       */
/************************************************************************/
void DataAccessKernelsLauncher::getData(
    GpuSettings const& gpuSettings,
    SimulationData const& data,
    int2 const& rectUpperLeft,
    int2 const& rectLowerRight,
    DataAccessTO const& dataTO)
{
    KERNEL_CALL_1_1(cudaClearDataTO, dataTO);
    KERNEL_CALL(cudaGetCellDataWithoutConnections, rectUpperLeft, rectLowerRight, data, dataTO);
    KERNEL_CALL(cudaResolveConnections, data, dataTO);
    KERNEL_CALL(cudaGetTokenData, data, dataTO);
    KERNEL_CALL(cudaGetParticleData, rectUpperLeft, rectLowerRight, data, dataTO);
}

void DataAccessKernelsLauncher::getSelectedData(
    GpuSettings const& gpuSettings,
    SimulationData const& data,
    bool includeClusters,
    DataAccessTO const& dataTO)
{
    KERNEL_CALL_1_1(cudaClearDataTO, dataTO);
    KERNEL_CALL(cudaGetSelectedCellDataWithoutConnections, data, includeClusters, dataTO);
    KERNEL_CALL(cudaResolveConnections, data, dataTO);
    KERNEL_CALL(cudaGetTokenData, data, dataTO);
    KERNEL_CALL(cudaGetSelectedParticleData, data, dataTO);
}

void DataAccessKernelsLauncher::getInspectedData(
    GpuSettings const& gpuSettings,
    SimulationData const& data,
    InspectedEntityIds entityIds,
    DataAccessTO const& dataTO)
{
    KERNEL_CALL_1_1(cudaClearDataTO, dataTO);
    KERNEL_CALL(cudaGetInspectedCellDataWithoutConnections, entityIds, data, dataTO);
    KERNEL_CALL(cudaResolveConnections, data, dataTO);
    KERNEL_CALL(cudaGetTokenData, data, dataTO);
    KERNEL_CALL(cudaGetInspectedParticleData, entityIds, data, dataTO);
}

void DataAccessKernelsLauncher::getOverlayData(
    GpuSettings const& gpuSettings,
    SimulationData const& data,
    int2 rectUpperLeft,
    int2 rectLowerRight,
    DataAccessTO dataTO)
{
    KERNEL_CALL_1_1(cudaClearDataTO, dataTO);
    KERNEL_CALL(cudaGetOverlayData, rectUpperLeft, rectLowerRight, data, dataTO);
}

void DataAccessKernelsLauncher::addData(GpuSettings const& gpuSettings, SimulationData data, DataAccessTO dataTO, bool selectData)
{
    KERNEL_CALL_1_1(cudaPrepareSetData, data);
    KERNEL_CALL(cudaAdaptNumberGenerator, data.numberGen, dataTO);
    KERNEL_CALL(cudaCreateDataFromTO, data, dataTO, selectData);
    _garbageCollector.cleanupAfterDataManipulation(gpuSettings, data);
    if (selectData) {
        KERNEL_CALL_1_1(cudaRolloutSelection, data);
    }

    hipDeviceSynchronize();
    CHECK_FOR_CUDA_ERROR(hipGetLastError());
}

void DataAccessKernelsLauncher::clearData(GpuSettings const& gpuSettings, SimulationData data) {}
